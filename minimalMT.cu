#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define NTHREADS 256
#define NBLOCKS 64
#define RNG_MAX 4294967295 
		
__global__ void generate_kernel(hiprandStateMtgp32 *state, double *result)
{
    int id = threadIdx.x + blockIdx.x * NTHREADS;
    /* Generate pseudo-random unsigned ints */
    result[id] = (double) hiprand(&state[blockIdx.x]) / RNG_MAX;
}



int main()
{
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;

	double *dev_results;

  /* Allocate space for prng states on device */
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, NBLOCKS * sizeof(hiprandStateMtgp32)));
 

  /* Setup MTGP prng states */
  
  /* Allocate space for MTGP kernel parameters */
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
  
  /* Reformat from predefined parameter sets to kernel format, */
  /* and copy kernel parameters to device memory               */
  CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
  
  /* Initialize one state per thread block */
  CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, 
              mtgp32dc_params_fast_11213, devKernelParams, NBLOCKS, 1234));
  
	CURAND_CALL(hipMalloc((void**)&dev_results, NBLOCKS * NTHREADS * sizeof(double)));

  /* State setup is complete */
  
	generate_kernel<<<NBLOCKS,NTHREADS>>>(devMTGPStates, dev_results);

	double host_results[NBLOCKS * NTHREADS];

  CUDA_CALL(hipMemcpy(host_results, dev_results, NBLOCKS * NTHREADS * sizeof(double), hipMemcpyDeviceToHost));

	for(int i = 0; i < NBLOCKS * NTHREADS; i++) { printf("host_result[%d] = %f\n", i, host_results[i]); }

  CUDA_CALL(hipFree(devMTGPStates));
  CUDA_CALL(hipFree(dev_results));

 	return EXIT_SUCCESS;
}
