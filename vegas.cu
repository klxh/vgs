#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>


#define vegas_cycles 1 // numero di iterazioni vegas
#define dim 2 // numero di variabili della funzione integranda
#define Nc 10 // numero di suddivisioni degli intervalli di integrazione
#define NTHREADS 16
#define NBLOCKS 4
#define RNG_MAX 4294967295 
#define DEFAULT_SEED 5234

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

		
__global__ void generate_kernel(hiprandStateMtgp32 *state, double *result)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Generate pseudo-random unsigned ints */
    result[id] = (double) hiprand(&state[blockIdx.x]) / RNG_MAX;
}

__global__ void integration(hiprandStateMtgp32 *state, float *grid, float *spacings, float *I, float *E)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; 
}

// definizione della funzione integranda
__device__ float f(float r[dim])
{
	return r[0] + r[1];	
}


int main ()
{
	int seed = DEFAULT_SEED;

	// definizione della griglia di integrazione e inizializzazione con equispaziature
	float grid[dim * (Nc + 1)];
	for(int i = 0; i < dim; i++)
	{
		grid[i * (Nc + 1)] = 0;
		printf("grid[%d][%d] = %f\n", i, 0, grid[i * (Nc + 1)]);
		for(int j = 1; j < Nc + 1; j++)
		{
			grid[i * (Nc + 1) + j] = grid[i * (Nc + 1) + j - 1] + 1./Nc;
		  printf("grid[%d][%d] = %f\n", i, j, grid[i * (Nc + 1) + j]);
		}
	}

	// definizione della griglia delle spaziature
	float spacings[dim * Nc];
	for(int i = 0; i < dim * Nc; i++) { spacings[i] = 1./Nc; }

	// set up per generazione di numeri random
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
	
	double *dev_results;

  /* Allocate space for prng states on device */
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, NBLOCKS * sizeof(hiprandStateMtgp32)));
 

  /* Setup MTGP prng states */
  
  /* Allocate space for MTGP kernel parameters */
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
  
  /* Reformat from predefined parameter sets to kernel format, */
  /* and copy kernel parameters to device memory               */
  CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
  
  /* Initialize one state per thread block */
  CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, 
              mtgp32dc_params_fast_11213, devKernelParams, NBLOCKS, seed));
  
	CURAND_CALL(hipMalloc((void**)&dev_results, NBLOCKS * NTHREADS * sizeof(double)));

  /* State setup is complete */
  
	generate_kernel<<<NBLOCKS,NTHREADS>>>(devMTGPStates, dev_results);

	double host_results[NBLOCKS * NTHREADS];

  CUDA_CALL(hipMemcpy(host_results, dev_results, NBLOCKS * NTHREADS * sizeof(double), hipMemcpyDeviceToHost));

	for(int i = 0; i < NBLOCKS * NTHREADS; i++) { printf("host_result[%d] = %f\n", i, host_results[i]); }

	// riprendo il discorso principale

	float *dev_grid;
	CUDA_CALL(hipMalloc((void**)&dev_grid, dim * (Nc + 1) * sizeof(float)));
	CUDA_CALL(hipMemcpy(dev_grid, grid,  dim * (Nc + 1) * sizeof(float), hipMemcpyHostToDevice));

	float *dev_spacings;
	CUDA_CALL(hipMalloc((void**)&dev_spacings, dim * Nc * sizeof(float)));
	CUDA_CALL(hipMemcpy(dev_spacings, spacings,  dim * Nc * sizeof(float), hipMemcpyHostToDevice));

	float I[vegas_cycles]; 
	float E[vegas_cycles];

	float *dev_I, *dev_E;

	CUDA_CALL(hipMalloc((void**)&dev_I, vegas_cycles * sizeof(float)));
	CUDA_CALL(hipMemset(dev_I, 0, vegas_cycles * sizeof(float)));

	CUDA_CALL(hipMalloc((void**)&dev_E, vegas_cycles * sizeof(float)));
	CUDA_CALL(hipMemset(dev_E, 0, vegas_cycles * sizeof(float)));

	// cicli vegas
	for (int it = 0; it < vegas_cycles; it++)
	{
		printf("\n###### ITERAZIONE VEGAS %d ######\n\n", it);

		integration<<<NBLOCKS,NTHREADS>>>(devMTGPStates, dev_grid, dev_spacings, dev_I, dev_E);
	}
	
	CUDA_CALL(hipMemcpy(I, dev_I, vegas_cycles * sizeof(float), hipMemcpyDeviceToHost));
	for(int i = 0; i < vegas_cycles; i++) { printf("I[%d] = %f\n", i, I[i]); }

  CUDA_CALL(hipFree(devMTGPStates));
  CUDA_CALL(hipFree(dev_results));
  CUDA_CALL(hipFree(dev_grid));
  CUDA_CALL(hipFree(dev_spacings));
  CUDA_CALL(hipFree(dev_I));
  CUDA_CALL(hipFree(dev_E));
 
 	return EXIT_SUCCESS;
}


